#include <stdlib.h>
#include <stdio.h>

#include "support.h"

void InitialGrid(int *grid, int  height, int width)
{
        int i, j;
        for(i=0;i<height;i++)
                for(j=0;j<width;j++)
                        grid[0*width*height+i*width+j]=grid[1*width*height+i*width+j]=0;
}


void GiveLife(int flag,int n, int *grid,int  height, int width)
{
        int i;
        for(i=0;i<n;i++)
                grid[flag*height*width+(rand()%height)*width+rand()%width]=1;
}

Matrix allocateMatrix(unsigned height, unsigned width)
{
	Matrix mat;
	mat.height = height;
	mat.width = mat.pitch = width;
	mat.elements = (float*)malloc(height*width*sizeof(float));
	if(mat.elements == NULL) FATAL("Unable to allocate host");

	return mat;
}

void initMatrix(Matrix mat)
{
    for (unsigned int i=0; i < mat.height*mat.width; i++) {
        mat.elements[i] = (rand()%100)/100.00;
    }
}

int IsLocationValid_CPU(int x, int y,int width, int height)
{
        if(x<0||y<0||x>=height||y>=width) return 0;
        else return 1;
}


int CountNeighbors_CPU(int flag,int x, int y, int  width, int height,int *grid)
{
        int count=0;
        int i, j;
        int range =3;
        for(i=-(range/2);i<=(range/2);i++)
        {
                for(j=-(range/2);j<=(range/2);j++)
                {
                        if(i==0&&j==0) continue;
                        if(IsLocationValid_CPU(x+i,y+j, width,height)==0) continue;
                        if(grid[flag*width*height+(x+i)*width +y+j]==1) count++;
                }
        }
        return count;
}

void GameofLife_CPU( int *grid, int width, int height, int nowGrid)
{
	int count;
    for(int i=0;i<height;i++){
        for(int j=0;j<width;j++){
            count=CountNeighbors_CPU(nowGrid,i,j,width,height,grid);
            if(grid[nowGrid*width*height+i*width+j]==0){
                if(count==3) grid[(1-nowGrid)*width*height+i*width+j]=1;
                else grid[(1-nowGrid)*width*height+i*width+j]=0;
            } else {
                if(count<=1||count>=4) grid[(1-nowGrid)*width*height+i*width+j]=0;
                else grid[(1-nowGrid)*width*height+i*width+j]=1;
            }
        }
    }
}


Matrix allocateDeviceMatrix(unsigned height, unsigned width)
{
	Matrix mat;
	hipError_t cuda_ret;

	mat.height = height;
	mat.width = mat.pitch = width;
	cuda_ret = hipMalloc((void**)&(mat.elements), height*width*sizeof(float));
	if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");

	return mat;
}

void copyToDeviceMatrix(Matrix dst, Matrix src)
{
	hipError_t cuda_ret;
	cuda_ret = hipMemcpy(dst.elements, src.elements, src.height*src.width*sizeof(float), hipMemcpyHostToDevice);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy to device");
}

void copyFromDeviceMatrix(Matrix dst, Matrix src)
{
	hipError_t cuda_ret;
	cuda_ret = hipMemcpy(dst.elements, src.elements, src.height*src.width*sizeof(float), hipMemcpyDeviceToHost);
	if(cuda_ret != hipSuccess) FATAL("Unable to copy from device");
}

void verify(int *GPU_result, int *CPU_result, int height, int width) {

      for(int i=0;i<2*width*height; i++)
      {
	if(GPU_result[i]!=CPU_result[i])
	{
		printf("TEST FAILED\n\n");
        	exit(0);		
	}
      }			
		
  printf("TEST PASSED\n\n");

}

void freeMatrix(Matrix mat)
{
	free(mat.elements);
	mat.elements = NULL;
}

void freeDeviceMatrix(Matrix mat)
{
	hipFree(mat.elements);
	mat.elements = NULL;
}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

