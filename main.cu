#include <stdio.h>
#include <iostream>
#include "support.h"
#include "kernel.cu"

int main(int argc, char* argv[])
{
    Timer timer;

    time_t t;
    
    /* Intializes random number generator */
    srand((unsigned) time(&t));    

    // Initialize host variables ----------------------------------------------

    printf("\nSetting up the problem..."); fflush(stdout);
    startTime(&timer);

    hipError_t cuda_ret;
    int  height, width;
    dim3 dim_grid, dim_block;

	/* Read image dimensions */
    if (argc == 1) {
        height = 1400;
	width =1400;
    } else if (argc == 2) {
        height= atoi(argv[1]);
	width= atoi(argv[1]);
    } else if (argc == 3) {
        height = atoi(argv[1]);
        width = atoi(argv[2]);
    }else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./gameoflife          # Matrix is 1400 x 1400"
           "\n    Usage: ./convolution <m>      # Matrix is m*m"
	   "\n    Usage: ./convolution <m> <n>  # Image is m x n"
           "\n");
        exit(0);
    }
	/* Allocate host memory */
	int *grid=new int [height*width*2];
	int *Ggrid_result=new int [height*width*2];
	/* Initialize Matrix */
	InitialGrid(grid,height,width);
	GiveLife(0,height*width/2,grid,height,width);


    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Allocate device variables ----------------------------------------------

    printf("Allocating device variables..."); fflush(stdout);
    startTime(&timer);
	int *GPUgrid;

	long long int size=sizeof(int)*2*width*height;
	(hipMalloc((void**) &GPUgrid, size));

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy host variables to device ------------------------------------------

    printf("Copying data from host to device..."); fflush(stdout);
    startTime(&timer);

	cuda_ret =(hipMemcpy(GPUgrid,grid,size,hipMemcpyHostToDevice));
	if(cuda_ret != hipSuccess) FATAL("Unable to copy to constant memory");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Launch kernel ----------------------------------------------------------
    printf("Launching kernel..."); fflush(stdout);
    startTime(&timer);

    //INSERT CODE HERE
	dim_block.x = BLOCK_SIZE; dim_block.y = BLOCK_SIZE; dim_block.z = 1;
	dim_grid.x = width/TILE_SIZE;
	if(width%TILE_SIZE != 0) dim_grid.x++;
	dim_grid.y = height/TILE_SIZE;
	if(height%TILE_SIZE != 0) dim_grid.y++;
	dim_grid.z = 1;
	
	int select =0;
	for(int m=0;m<ITERATION;m++){
        	GameofLife<<<dim_grid, dim_block>>>(GPUgrid,select,width,height);
         	select=1-select;
        }
	cuda_ret = hipDeviceSynchronize();
	if(cuda_ret != hipSuccess) FATAL("Unable to launch/execute kernel");

    hipDeviceSynchronize();
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    // Copy device variables from host ----------------------------------------

    printf("Copying data from device to host...\n"); fflush(stdout);
    startTime(&timer);

    hipMemcpy(Ggrid_result,GPUgrid,size,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

   //CPU -----------------------------------------------------------------------

	int nowGrid=0;
        for(int n=0;n<ITERATION;n++)
        {
		GameofLife_CPU( grid, width, height,nowGrid);
		nowGrid=1-nowGrid;
		
      	}

// Verify correctness -----------------------------------------------------
	printf("Verifying..."); fflush(stdout);
	verify(Ggrid_result,grid,height,width);

// Free memory ------------------------------------------------------------
	 hipFree(GPUgrid);

	 delete [] grid;		
	 delete [] Ggrid_result;
	 return 0;
}
