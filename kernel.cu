#include "hip/hip_runtime.h"
__global__ void GameofLife(int *GPUgrid,  int select, int width, int height){


     int outRow= blockIdx.x*TILE_SIZE + threadIdx.x;
     int outCol= blockIdx.y*TILE_SIZE + threadIdx.y;
     int count=0;

     __shared__  int GPUgrid_shin[BLOCK_SIZE][BLOCK_SIZE];

     int inRow = outRow-1;
     int inCol = outCol-1;

     if(inRow >= 0 && inRow < width && inCol >= 0 && inCol < height){
        GPUgrid_shin[threadIdx.y][threadIdx.x]=GPUgrid[select*width*height+inCol*width+inRow];
     } else{
        GPUgrid_shin[threadIdx.y][threadIdx.x] = 0;
    }
       __syncthreads();

    int tidx=threadIdx.x+1;
    int tidy=threadIdx.y+1;

    if(threadIdx.x < TILE_SIZE && threadIdx.y < TILE_SIZE && outRow<width && outCol<height) {
        count=GPUgrid_shin[tidy+1][tidx]+GPUgrid_shin[tidy-1][tidx]+GPUgrid_shin[tidy][tidx+1]+GPUgrid_shin[tidy][tidx-1]+GPUgrid_shin[tidy+1][tidx+1]+GPUgrid_shin[tidy-1][tidx-1]+GPUgrid_shin[tidy-1][tidx+1]+GPUgrid_shin[ tidy+1][tidx-1];
        if(GPUgrid_shin[tidy][tidx]==0){
            if(count==3) {
                GPUgrid[(1-select)*height*width+outCol*width+outRow]=1;
            } else {
                GPUgrid[(1-select)*height*width+outCol*width+outRow]=0;
            }
        } else {
            if(count<=1||count>=4) {
                GPUgrid[(1-select)*height*width+outCol*width+outRow]=0;
            } else {
                GPUgrid[(1-select)*height*width+outCol*width+outRow]=1;
            }
        }
    }
}




























